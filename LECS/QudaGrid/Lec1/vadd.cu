#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#include <sys/time.h>

struct timeval t1, t2;

#define N (32*1024*1024)
#define MAX_ERR 1e-6

// single thread and block
__global__ void vector_add(float *out, float *a, float *b, int n)
{
    for(size_t i = 0; i < n; i ++){
        out[i] = a[i] + b[i];
    }
}

// multi block
__global__ void vadd_block(float *out, float *a, float *b, int n)
{
      out[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x] ;
}

// multi thread
__global__ void vadd_thread(float *out, float *a, float *b, int n)
{
      out[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x] ;
}

// combined
__global__ void vadd_blth(float *out, float *a, float *b, int n)
{
     // unique id
     int i = blockIdx.x * blockDim.x + threadIdx.x;
     if( i < n ) {
        out[i] = a[i] + b[i] ;
     }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    hipHostMalloc( &a , N*sizeof(float) , hipHostMallocDefault) ;
    hipHostMalloc( &b , N*sizeof(float) , hipHostMallocDefault) ;
    hipHostMalloc( &out , N*sizeof(float) , hipHostMallocDefault) ;

    // Initialize host arrays
    for(size_t i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    gettimeofday(&t1, 0);	       

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    hipDeviceSynchronize() ;
    gettimeofday(&t2, 0) ;
    double time = (t2.tv_usec-t1.tv_usec)/1000.0;
    printf("Time to copy:  %3.1f ms \n", time);

    for( int th = 1 ; th<2048 ; th*=2 ) {
        if( N%th == 1 ) continue ;
        gettimeofday(&t1, 0);
    	// Executing kernel 
    	//vector_add<<<1,1>>>(d_out, d_a, d_b, N);
    	//vadd_block<<<N,1>>>(d_out, d_a, d_b, N);
    	//vadd_thread<<<1,N>>>(d_out, d_a, d_b, N);
    	vadd_blth<<<N/th,th>>>(d_out, d_a, d_b, N);
    	gettimeofday(&t2, 0) ;
    	time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    	printf("Time to execute th = %d :  %e ms \n", th , time);
    }
    hipDeviceSynchronize() ;
    
    gettimeofday(&t1, 0) ;

    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    gettimeofday(&t2, 0) ;
    time = (t2.tv_usec-t1.tv_usec)/1000.0;
    printf("Time to copy:  %3.1f ms \n", time);

    // Verification
    for(size_t i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    hipHostFree(a) ;
    hipHostFree(b) ;	
    hipHostFree(out) ;
}
