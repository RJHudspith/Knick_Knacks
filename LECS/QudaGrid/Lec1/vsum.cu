#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#include <sys/time.h>

struct timeval t1, t2;

#define N (32*1024*1024)
#define MAX_ERR 1e-6

/*
// single block sum over threads
__global__ void vsum(float *out, float *a, int n)
{
    float sum = 0 ;
    for( int i = 0 ; i < n ; i++ ) {
        sum += a[i] ;
    }
    __syncthreads() ;
    out[0] = sum ;
}
*/

// multi-block and multithread
__global__ void vsum(float *out, float *a, int n)
{
    extern __shared__ float sdata[] ; // shared memory
    int tid = threadIdx.x ;
    int idx = blockIdx.x*blockDim.x+tid ;
    sdata[tid] = a[idx] ; // global to local
    __syncthreads() ;
    for( int s = blockDim.x/2 ; s > 0 ; s>>=1 ) {
    	 if( tid < s ) {
	     sdata[tid] += sdata[tid+s] ;
	 }
	     __syncthreads() ;
    }
    if( tid == 0 ) out[blockIdx.x] = sdata[0] ;
}

__device__ inline void wred( volatile float *sdata , const int tid )
{
   sdata[tid] += sdata[tid+32] ;
   sdata[tid] += sdata[tid+16] ;
   sdata[tid] += sdata[tid+8] ;
   sdata[tid] += sdata[tid+4] ;
   sdata[tid] += sdata[tid+2] ;
   sdata[tid] += sdata[tid+1] ;
}

// multi-block and multithread
__global__ void vsum2(float *out, float *a, int n)
{
    extern __shared__ float sdata[] ; // shared memory
    int tid = threadIdx.x ;
    int idx = blockIdx.x*blockDim.x+tid ;
    sdata[tid] = a[idx] ; // global to local
    __syncthreads() ;
    for( int s = blockDim.x/2 ; s > 32 ; s>>=1 ) {
    	 if( tid < s ) {
	     sdata[tid] += sdata[tid+s] ;
	 }
	     __syncthreads() ;
    }
    if( tid < 32 ) wred( sdata , tid ) ;
    if( tid == 0 ) out[blockIdx.x] = sdata[0] ;
}

int main(){
    float *a ;
    float *d_a, *d_out; 

    // Allocate host memory
    hipHostMalloc( &a , sizeof(float) * N, hipHostMallocDefault);

    // Initialize host arrays
    for(size_t i = 0; i < N; i++){
        a[i] = 1.0f;
    }
    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);

    gettimeofday(&t1, 0);	       

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);

	 hipDeviceSynchronize() ;   
    gettimeofday(&t2, 0) ;
    double time = (t2.tv_usec-t1.tv_usec)/1000.;
    printf("Time to copy:  %3.1f ms \n", time);

    for( int th = 1 ; th < 512 ; th*=2 ) {
    	 if( N%th != 0 ) continue ;
	 const int Block = N/th ;
    	 float *out ; 
	 hipHostMalloc( &out , Block*sizeof(float) , hipHostMallocDefault); 
         hipMalloc( (void**)&d_out, sizeof(float) * Block);

	 gettimeofday(&t1, 0);	       
	 vsum2<<<Block,th>>>(d_out, d_a, N);
	 //vsum<<<Block,th>>>(d_out, d_a, N);
	 gettimeofday(&t2, 0) ;
	 double time = (t2.tv_usec-t1.tv_usec)/1000. ;
    	 printf("Time to execute: Nth %d  %e ms \n", th , time);

	 hipDeviceSynchronize() ;
	 
	 gettimeofday(&t1, 0);	       
	 // Transfer data back to host memory
    	 hipMemcpy(out, d_out, sizeof(float) * Block, hipMemcpyDeviceToHost);
	 gettimeofday(&t2, 0) ;
	 time = (t2.tv_usec-t1.tv_usec)/1000. ;
    	 printf("Copy back  %e ms \n", time);

    	 // traditional sum outside
    	 float sum = 0 ;
    	 for( int i = 0 ; i < Block ; i++ ) {
    	      sum += out[i] ;
    	 }
    	 printf( "Sum %f N %d\n" , sum , N ) ;

         hipFree(d_out);
    	 hipHostFree(out) ;
    }

    // Deallocate device memory
    hipFree(d_a);

    // Deallocate host memory
    hipHostFree(a);
}
